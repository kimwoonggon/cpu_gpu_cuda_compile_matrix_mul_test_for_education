#include "hip/hip_runtime.h"
#include "gpu_mul.h"

__global__ void matrixMultiplyKernel(float *a, float *b, float *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        float sum = 0.0f;
        for(int k = 0; k < N; ++k) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

void gpuMatrixMultiply(float *a, float *b, float *c, int N) {
    float *dev_a, *dev_b, *dev_c;
    
    hipMalloc((void**)&dev_a, N*N*sizeof(float));
    hipMalloc((void**)&dev_b, N*N*sizeof(float));
    hipMalloc((void**)&dev_c, N*N*sizeof(float));
    
    hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
    
    hipMemcpy(c, dev_c, N*N*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
